
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void use_local_memory_GPU(float in)
{
    float f;    // variable "f" is in local memory and private to each thread
    f = in;     // parameter "in" is in local memory and private to each thread
}


__global__ void use_global_memory_GPU(float *array)
{
    array[threadIdx.x] = 2.0f * (float) threadIdx.x;
}


__global__ void use_shared_memory_GPU(float *array)
{
    int i, index = threadIdx.x;
    float average, sum = 0.0f;

    __shared__ float sh_arr[128];

    sh_arr[index] = array[index];

    __syncthreads();    // ensure all the writes to shared memory have completed

    for (i=0; i<index; i++) { sum += sh_arr[i]; }
    average = sum / (index + 1.0f);

	printf("Thread id = %d\t Average = %f\n",index,average);
    if (array[index] > average) { array[index] = average; }

    sh_arr[index] = 3.14;
}

int main(int argc, char **argv)
{
    use_local_memory_GPU<<<1, 128>>>(2.0f);

    float h_arr[128];   // convention: h_ variables live on host
    float *d_arr;       // convention: d_ variables live on device (GPU global mem)

    hipMalloc((void **) &d_arr, sizeof(float) * 128);
    hipMemcpy((void *)d_arr, (void *)h_arr, sizeof(float) * 128, hipMemcpyHostToDevice);
    use_global_memory_GPU<<<1, 128>>>(d_arr);  // modifies the contents of array at d_arr
    hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128, hipMemcpyDeviceToHost);


    use_shared_memory_GPU<<<1, 128>>>(d_arr); 
    hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    return 0;
}

